#include "hip/hip_runtime.h"
#include <stdio.h>
#include <costmap_2d/cuda_static_layer.h>

#define TPB 512
#define NO_INFORMATION 255

__global__ void updateWithTrueOverwriteKernel(unsigned char *master,unsigned char *costmap,unsigned long size, int min_i, int min_j, int max_i, int max_j, int span)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	int deltaj=id/(max_j-min_j);	//Row
	int deltai=id%(max_i-min_i);	//Coloum
	int j=min_j+deltaj;
	int i=min_i+deltai;
	int index=span*j+i;
	if(index<size)
		master[index]=costmap[index];
}

__global__ void updateWithMaxKernel(unsigned char *master,unsigned char *costmap,unsigned long size, int min_i, int min_j, int max_i, int max_j, int span)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	int deltaj=id/(max_j-min_j);	//Row
	int deltai=id%(max_i-min_i);	//Coloum
	int j=min_j+deltaj;
	int i=min_i+deltai;
	int index=span*j+i;
	if(index<size)
	{
		if(costmap[i]==NO_INFORMATION)
			return;
		unsigned char oldCost=master[index];
		if(oldCost==NO_INFORMATION||oldCost<costmap[index])
			master[index]=costmap[index];
	}
}

void costmap_2d::cuda::updateWithTrueOverwrite(costmap_2d::Costmap2D& master_grid, int min_i, int min_j, int max_i, int max_j, unsigned char *costmap_)
{
	//Unnecessary. Has been checked outside.
	/*********
	if (!enabled_)
    	return;
	 *********/
	unsigned char *master = master_grid.getCharMap();
	unsigned int span = master_grid.getSizeInCellsX();
	unsigned long size=master_grid.getSizeInCellsX()*master_grid.getSizeInCellsY();

	unsigned char *cuda_master=NULL;
	unsigned char *cuda_costmap=NULL;
	hipMalloc(&cuda_master,sizeof(unsigned char)*size);
	hipMalloc(&cuda_costmap,sizeof(unsigned char)*size);

	hipMemcpy(cuda_master,master,sizeof(unsigned char)*size,hipMemcpyHostToDevice);
	hipMemcpy(cuda_costmap,costmap_,sizeof(unsigned char)*size,hipMemcpyHostToDevice);

	updateWithTrueOverwriteKernel<<<(size+TPB-1)/TPB,TPB>>>(cuda_master,cuda_costmap,size,min_i,min_j,max_i,max_j,span);

	hipMemcpy(master,cuda_master,sizeof(unsigned char)*size,hipMemcpyDeviceToHost);
	hipFree(cuda_master);
	hipFree(cuda_costmap);
}

void costmap_2d::cuda::updateWithMax(costmap_2d::Costmap2D& master_grid, int min_i, int min_j, int max_i, int max_j, unsigned char *costmap_)
{
	//Unnecessary. Has been checked outside.
	/*********
	if (!enabled_)
    	return;
	 *********/
	unsigned char *master = master_grid.getCharMap();
	unsigned int span = master_grid.getSizeInCellsX();
	unsigned long size=master_grid.getSizeInCellsX()*master_grid.getSizeInCellsY();

	unsigned long sizeToUpdate=(max_j-min_j)*(max_i-min_i);

	unsigned char *cuda_master=NULL;
	unsigned char *cuda_costmap=NULL;
	hipMalloc(&cuda_master,sizeof(unsigned char)*size);
	hipMalloc(&cuda_costmap,sizeof(unsigned char)*size);

	hipMemcpy(cuda_master,master,sizeof(unsigned char)*size,hipMemcpyHostToDevice);
	hipMemcpy(cuda_costmap,costmap_,sizeof(unsigned char)*size,hipMemcpyHostToDevice);

	updateWithMaxKernel<<<(sizeToUpdate+TPB-1)/TPB,TPB>>>(cuda_master,cuda_costmap,size,min_i,min_j,max_i,max_j,span);

	hipMemcpy(master,cuda_master,sizeof(unsigned char)*size,hipMemcpyDeviceToHost);
	hipFree(cuda_master);
	hipFree(cuda_costmap);
}
                  