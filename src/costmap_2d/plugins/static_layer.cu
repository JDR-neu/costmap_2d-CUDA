#include "hip/hip_runtime.h"
#include <stdio.h>
#include <tf/tf.h>
#include <costmap_2d/cuda_static_layer.h>

#define TPB 512
#define NO_INFORMATION 255

struct CostMapParameters
{
	double origin_x;
	double origin_y;
	double resolution;
	int span;
};

__global__ void updateWithTrueOverwriteKernel(unsigned char *master,unsigned char *costmap,unsigned long size, int min_i, int min_j, int max_i, int max_j, int span)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	int deltaj=id/(max_j-min_j);	//Row
	int deltai=id%(max_i-min_i);	//Coloum
	int j=min_j+deltaj;
	int i=min_i+deltai;
	int index=span*j+i;
	if(index<size)
		master[index]=costmap[index];
}

__global__ void updateWithMaxKernel(unsigned char *master,unsigned char *costmap,unsigned long size, int min_i, int min_j, int max_i, int max_j, int span)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	int deltaj=id/(max_j-min_j);	//Row
	int deltai=id%(max_i-min_i);	//Coloum
	int j=min_j+deltaj;
	int i=min_i+deltai;
	int index=span*j+i;
	if(index<size)
	{
		if(costmap[i]==NO_INFORMATION)
			return;
		unsigned char oldCost=master[index];
		if(oldCost==NO_INFORMATION||oldCost<costmap[index])
			master[index]=costmap[index];
	}
}

__global__ void rollingUpdateCostsKernel(unsigned char *master, CostMapParameters masterParams,
	unsigned char *costmap, CostMapParameters costmapParams, tf::TransformData serializedTF,
	int min_i, int min_j, int max_i, int max_j, bool use_maximum)
{
}

void costmap_2d::cuda::updateWithTrueOverwrite(costmap_2d::Costmap2D& master_grid, int min_i, int min_j, int max_i, int max_j, unsigned char *costmap_)
{
	//Unnecessary. Has been checked outside.
	/*********
	if (!enabled_)
    	return;
	 *********/
	unsigned char *master = master_grid.getCharMap();
	unsigned int span = master_grid.getSizeInCellsX();
	unsigned long size=master_grid.getSizeInCellsX()*master_grid.getSizeInCellsY();

	unsigned long sizeToUpdate=(max_j-min_j)*(max_i-min_i);

	unsigned char *cuda_master=NULL;
	unsigned char *cuda_costmap=NULL;
	hipMalloc(&cuda_master,sizeof(unsigned char)*size);
	hipMalloc(&cuda_costmap,sizeof(unsigned char)*size);

	hipMemcpy(cuda_master,master,sizeof(unsigned char)*size,hipMemcpyHostToDevice);
	hipMemcpy(cuda_costmap,costmap_,sizeof(unsigned char)*size,hipMemcpyHostToDevice);

	updateWithTrueOverwriteKernel<<<(sizeToUpdate+TPB-1)/TPB,TPB>>>(cuda_master,cuda_costmap,size,min_i,min_j,max_i,max_j,span);

	hipMemcpy(master,cuda_master,sizeof(unsigned char)*size,hipMemcpyDeviceToHost);
	hipFree(cuda_master);
	hipFree(cuda_costmap);
}

void costmap_2d::cuda::updateWithMax(costmap_2d::Costmap2D& master_grid, int min_i, int min_j, int max_i, int max_j, unsigned char *costmap_)
{
	//Unnecessary. Has been checked outside.
	/*********
	if (!enabled_)
    	return;
	 *********/
	unsigned char *master = master_grid.getCharMap();
	unsigned int span = master_grid.getSizeInCellsX();
	unsigned long size=master_grid.getSizeInCellsX()*master_grid.getSizeInCellsY();

	unsigned long sizeToUpdate=(max_j-min_j)*(max_i-min_i);

	unsigned char *cuda_master=NULL;
	unsigned char *cuda_costmap=NULL;
	hipMalloc(&cuda_master,sizeof(unsigned char)*size);
	hipMalloc(&cuda_costmap,sizeof(unsigned char)*size);

	hipMemcpy(cuda_master,master,sizeof(unsigned char)*size,hipMemcpyHostToDevice);
	hipMemcpy(cuda_costmap,costmap_,sizeof(unsigned char)*size,hipMemcpyHostToDevice);

	updateWithMaxKernel<<<(sizeToUpdate+TPB-1)/TPB,TPB>>>(cuda_master,cuda_costmap,size,min_i,min_j,max_i,max_j,span);

	hipMemcpy(master,cuda_master,sizeof(unsigned char)*size,hipMemcpyDeviceToHost);
	hipFree(cuda_master);
	hipFree(cuda_costmap);
}
 
void costmap_2d::cuda::static_layer::rollingUpdateCosts(costmap_2d::Costmap2D& master_grid, tf::StampedTransform tf, costmap_2d::Costmap2D *costmap, bool use_maximum, int min_i, int min_j, int max_i, int max_j)
{
	struct tf::TransformData serializedTF;
	tf.serialize(serializedTF);

	unsigned char *master = master_grid.getCharMap();
	unsigned char *costmap_grid = costmap->getCharMap();

	struct CostMapParameters masterParams,costmapParams;	
	masterParams.span = master_grid.getSizeInCellsX();
	masterParams.resolution = master_grid.getResolution();
	masterParams.origin_x = master_grid.getOriginX();
	masterParams.origin_y = master_grid.getOriginY();
	unsigned long master_size = master_grid.getSizeInCellsX()*master_grid.getSizeInCellsY();
	costmapParams.span = costmap->getSizeInCellsX();
	costmapParams.resolution = costmap->getResolution();
	masterParams.origin_x = costmap->getOriginX();
	masterParams.origin_y = costmap->getOriginY();
	unsigned long costmap_size=costmap->getSizeInCellsX()*costmap->getSizeInCellsY();

	unsigned long sizeToUpdate=(max_j-min_j)*(max_i-min_i);

	unsigned char *cuda_master=NULL;
	unsigned char *cuda_costmap=NULL;
	hipMalloc(&cuda_master,sizeof(unsigned char)*master_size);
	hipMalloc(&cuda_costmap,sizeof(unsigned char)*costmap_size);

	hipMemcpy(cuda_master,master,sizeof(unsigned char)*master_size,hipMemcpyHostToDevice);
	hipMemcpy(cuda_costmap,costmap->getCharMap(),sizeof(unsigned char)*costmap_size,hipMemcpyHostToDevice);

	rollingUpdateCostsKernel<<<(sizeToUpdate+TPB-1)/TPB,TPB>>>(cuda_master,masterParams,cuda_costmap,costmapParams,serializedTF,min_i,min_j,max_i,max_j,use_maximum);
}